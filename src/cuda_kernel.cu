#define _USE_MATH_DEFINES


#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>
#include <cmath>
#include <hip/hip_runtime_api.h>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__
void multibrot_kernel(
        unsigned int unroll,
        unsigned char *image,
        int width, int height, double ratio,
        int exponent, int iterations, double R, double eps,
        unsigned char borderR, unsigned char borderG, unsigned char borderB, double borderThickness,
        long normOrbitSkip, double normLightIntensity, double normLightAngle, double normLightHeight,
        unsigned char bgR, unsigned char bgG, unsigned char bgB,
        double kR, double kG, double kB, double kD,
        unsigned char internalBorderR, unsigned char internalBorderG, unsigned char internalBorderB,
        unsigned char internalCoreR, unsigned char internalCoreG, unsigned char internalCoreB,
        double internalK,
        double stripeDensity, double stripeLightIntensity,
        double zoom, double posX, double posY
) {
    unsigned int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int unrollIndex = 0; unrollIndex < unroll; unrollIndex++) {
        unsigned int currentIndex = threadIndex * unroll + unrollIndex;
        if (currentIndex >= width * height) {
            return;
        }

        //region Calculations
        double c_r = (((currentIndex % width - 1) - width / 2.) / (width * zoom)) * ratio + posX;
        double c_i = ((double) currentIndex / width - height / 2.) / (height * zoom) + posY;

        double z_r = c_r;
        double z_i = c_i;

        double last_z_r = 0;
        double last_z_i = 0;

        double dz_r = 1.;
        double dz_i = 0.;

        double dc_r = 1.;
        double dc_i = 0.;

        double dzdz_r = 0.;
        double dzdz_i = 0.;

        double dcdc_r = 0.;
        double dcdc_i = 0.;

        double dcdz_r = 0.;
        double dcdz_i = 0.;

        double p = 1.;

        double orbitCount = 0;

        double V = 0;

        long i;
        for (i = 0; i < iterations; i++) {
            double z2 = z_r * z_r + z_i * z_i;
            if (z2 > R * R) {
                V = log(z2) / p;
                break;
            }

            if (eps > 0 && dz_r * dz_r + dz_i * dz_i < eps * eps) {
                V = 0;
                break;
            }

            double dzdz_r_temp = 2 * ((z_r * dzdz_r - z_i * dzdz_i) + (dz_r * dz_r - dz_i * dz_i));
            dzdz_i = 2 * ((z_r * dzdz_i + z_i * dzdz_r) + (dz_r * dz_i + dz_i * dz_r));
            dzdz_r = dzdz_r_temp;

            double dcdc_r_temp = 2 * ((z_r * dcdc_r - z_i * dcdc_i) + (dc_r * dc_r - dc_i * dc_i));
            dcdc_i = 2 * ((z_r * dcdc_i + z_i * dcdc_r) + (dc_r * dc_i + dc_i * dc_r));
            dcdc_r = dcdc_r_temp;

            double dcdz_r_temp = 2 * ((z_r * dcdz_r - z_i * dcdz_i) + (dz_r * dc_r - dz_i * dc_i));
            dcdz_i = 2 * ((z_r * dcdz_i + z_i * dcdz_r) + (dc_r * dz_i + dc_i * dz_r));
            dcdz_r = dcdz_r_temp;

            double dz_r_temp = 2 * (z_r * dz_r - z_i * dz_i);
            dz_i = 2 * (z_r * dz_i + z_i * dz_r);
            dz_r = dz_r_temp;

            double dc_r_temp = 2 * (z_r * dc_r - z_i * dc_i) + 1;
            dc_i = 2 * (z_r * dc_i + z_i * dc_r);
            dc_r = dc_r_temp;

            p *= 2.;

            if (i >= normOrbitSkip) {
                orbitCount += 0.5 + 0.5 * sin(stripeDensity * atan2(last_z_i, last_z_r));
            }
            last_z_r = z_r;
            last_z_i = z_i;

            int esp = exponent;
            if (esp != 0) {
                if (esp < 0) {
                    esp = -esp;

                    double z_r_temp = z_r / (z_r * z_r + z_i * z_i);
                    z_i = -z_i / (z_r * z_r + z_i * z_i);
                    z_r = z_r_temp;
                }
                double z_esp_r = z_r;
                double z_esp_i = z_i;
                for (int e = 1; e < esp; e++) {
                    double z_esp_r_temp = (z_r * z_esp_r - z_i * z_esp_i);
                    z_esp_i = (z_esp_i * z_r + z_i * z_esp_r);
                    z_esp_r = z_esp_r_temp;
                }
                z_r = z_esp_r + c_r;
                z_i = z_esp_i + c_i;
            } else {
                z_r = 1.0;
                z_i = 0.0;
            }

        }
        // endregion

        if (V == 0) { // Inside!
            //region Interior distance estimation
            double u_r = (dzdz_r * dc_r - dzdz_i * dc_i);
            double u_i = (dzdz_r * dc_i + dzdz_i * dc_r);
            double v_r = 1 - dz_r;
            double v_i = -dz_i;

            double u_r_temp = (u_r * v_r + u_i * v_i) / (v_r * v_r + v_i * v_i);
            u_i = (u_i * v_r - u_r * v_i) / (v_r * v_r + v_i * v_i);
            u_r = u_r_temp;

            u_r = u_r + dcdz_r;
            u_i = u_i + dcdz_i;

            double d = (1. - (dz_r * dz_r + dz_i * dz_i)) / sqrt(u_r * u_r + u_i * u_i);
            //endregion

//        if (d < 1) {
//            image[currentIndex * 4] = 0;
//            image[currentIndex * 4 + 1] = (int) max(0., min(255., (255. * tanh(d))));
//            image[currentIndex * 4 + 1] = (unsigned char) (max(0., min(255., 0 + d * (255 - 0))));
//            image[currentIndex * 4 + 2] = 0;
//        } else {
//            image[currentIndex * 4] = 0;
//            image[currentIndex * 4 + 1] = 255;
//            image[currentIndex * 4 + 2] = 0;
//        }

            double mix = internalK > 0 ? log(d) / internalK : 1;
            if (mix < 1) {
                image[currentIndex * 4] = max(0., min(255., internalBorderR + mix * (internalCoreR - internalBorderR)));
                image[currentIndex * 4 + 1] = max(0., min(255., internalBorderG + mix * (internalCoreG - internalBorderG)));
                image[currentIndex * 4 + 2] = max(0., min(255., internalBorderB + mix * (internalCoreB - internalBorderB)));
            } else {
                image[currentIndex * 4] = internalCoreR;
                image[currentIndex * 4 + 1] = internalCoreG;
                image[currentIndex * 4 + 2] = internalCoreB;
            }
        } else { // Outside!
            //region Exterior distance estimation
            double rad = sqrt(z_r * z_r + z_i * z_i);
            double d = rad * 2. * log(rad) / sqrt(dc_r * dc_r + dc_i * dc_i);
            //endregion

            unsigned char tempR = bgR;
            unsigned char tempG = bgG;
            unsigned char tempB = bgB;

            //region Gradient Background Setup
            if (kR > 0) {
                tempR = (unsigned char) (max(0., min(255., tempR + (255. * (1 + cos(M_PI_2 * log(V) / (kR))) / 2. / kD))));
            }
            if (kG > 0) {
                tempG = (unsigned char) (max(0., min(255., tempG + (255. * (1 + cos(M_PI_2 * log(V) / (kG))) / 2. / kD))));
            }
            if (kB > 0) {
                tempB = (unsigned char) (max(0., min(255., tempB + (255. * (1 + cos(M_PI_2 * log(V) / (kB))) / 2. / kD))));
            }
            //endregion

            //region 3D Normal
            if (normLightIntensity != 1) {
                double vR = cos(normLightAngle * 2. * M_PI / 360.);
                double vI = sin(normLightAngle * 2. * M_PI / 360.);
                double lo = 0.5 * log(z_r * z_r + z_i * z_i);
                double conjR = ((1. + lo) * (dc_r * dc_r - dc_i * dc_i) - (lo) * (z_r * dcdc_r - z_i * dcdc_i));
                double conjI = ((1. + lo) * -(dc_r * dc_i + dc_i * dc_r) - (lo) * -(z_r * dcdc_i + z_i * dcdc_r));
                double uR = (z_r * dc_r - z_i * dc_i);
                double uI = (z_r * dc_i + z_i * dc_r);
                double newUR = (uR * conjR - uI * conjI);
                uI = (uR * conjI + uI * conjR);
                uR = newUR;
                newUR = uR / sqrt(uR * uR + uI * uI);
                uI = uI / sqrt(uR * uR + uI * uI);
                uR = newUR;
                double t = uR * vR + uI * vI + normLightHeight;
                t = t / (1. + normLightHeight);
                if (t < 0) {
                    t = 0;
                } else if (t > 1) {
                    t = 1;
                }
                double normShadowIntensity = 1 + (1 - normLightIntensity);
                tempR = (unsigned char) (max(0., min(255., tempR * normShadowIntensity)) +
                                         t * (max(0., min(255., tempR *
                                                                normLightIntensity)) -
                                              max(0., min(255., tempR *
                                                                normShadowIntensity))));
                tempG = (unsigned char) (max(0., min(255., tempG * normShadowIntensity)) +
                                         t * (max(0., min(255., tempG *
                                                                normLightIntensity)) -
                                              max(0., min(255., tempG *
                                                                normShadowIntensity))));
                tempB = (unsigned char) (max(0., min(255., tempB * normShadowIntensity)) +
                                         t * (max(0., min(255., tempB *
                                                                normLightIntensity)) -
                                              max(0., min(255., tempB *
                                                                normShadowIntensity))));
            }
            //endregion

            //region Stripe Average Colouring
            if (stripeLightIntensity != 1) {
                double lastOrbit = 0.5 + 0.5 * sin(stripeDensity * atan2(last_z_i, last_z_r));
                double smallCount = orbitCount - lastOrbit;
                orbitCount /= (double) i;
                smallCount /= (double) i - 1;
                double frac = -1. + log10(2.0 * log(R * R)) / log10(2.) -
                              log10(0.5 * log(last_z_r * last_z_r + last_z_i * last_z_i)) / log10(2.);
                double mix = frac * orbitCount + (1 - frac) * smallCount;
                if (mix < 0) {
                    mix = 0;
                } else if (mix > 1) {
                    mix = 1;
                }
                double stripeShadowIntensity = 1 + (1 - stripeLightIntensity);
                unsigned char stripeLightR = max(0., min(255., tempR * stripeLightIntensity));
                unsigned char stripeLightG = max(0., min(255., tempG * stripeLightIntensity));
                unsigned char stripeLightB = max(0., min(255., tempB * stripeLightIntensity));
                unsigned char stripeShadowR = max(0., min(255., tempR * stripeShadowIntensity));
                unsigned char stripeShadowG = max(0., min(255., tempG * stripeShadowIntensity));
                unsigned char stripeShadowB = max(0., min(255., tempB * stripeShadowIntensity));
                tempR = (unsigned char) (stripeShadowR + (mix * (stripeLightR - stripeShadowR)));
                tempG = (unsigned char) (stripeShadowG + (mix * (stripeLightG - stripeShadowG)));
                tempB = (unsigned char) (stripeShadowB + (mix * (stripeLightB - stripeShadowB)));
            }
            //endregion

            //region Border
            if (borderThickness > 0) {
                double tBorder = d / borderThickness;
                if (tBorder < 1) { // Border
                    tempR = (unsigned char) (borderR + tBorder * (tempR - borderR));
                    tempG = (unsigned char) (borderG + tBorder * (tempG - borderG));
                    tempB = (unsigned char) (borderB + tBorder * (tempB - borderB));
                }
            }
            //endregion

            image[currentIndex * 4] = tempR;
            image[currentIndex * 4 + 1] = tempG;
            image[currentIndex * 4 + 2] = tempB;
        }
    }
}

void multibrot(
        unsigned int unroll,
        unsigned int blockSize,
        unsigned char *rgb,
        int width, int height,
        int exponent, int iterations, double R, double eps,
        unsigned char borderR, unsigned char borderG, unsigned char borderB, double borderThickness,
        long normOrbitSkip, double normLightIntensity, double normLightAngle, double normLightHeight,
        unsigned char bgR, unsigned char bgG, unsigned char bgB,
        double kR, double kG, double kB, double kD,
        unsigned char internalBorderR, unsigned char internalBorderG, unsigned char internalBorderB,
        unsigned char internalCoreR, unsigned char internalCoreG, unsigned char internalCoreB, double internalK,
        double stripeDensity, double stripeLightIntensity,
        double zoom, double posX, double posY) {
    hipProfilerStart();

    //region Setup
    cout << "Setting up..." << endl;
    double ratio = (double) width / height;
    unsigned int size = width * height;

    unsigned char *imageHost;
    imageHost = (unsigned char *) malloc(4 * size * sizeof(unsigned char));
    unsigned char *imageDevice;
    gpuErrchk(hipMallocManaged(&imageDevice, 4 * size * sizeof(unsigned char)));

    int suggestedBlockSize;
    int minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &suggestedBlockSize, multibrot_kernel, 0, 4 * size);
    cout << "Suggested BlockSize: " << suggestedBlockSize << endl << "Min GridSize: " << minGridSize << endl;

    int gridSize = (size + blockSize - 1) / blockSize / unroll;
    cout << "BlockSize: " << blockSize << endl << "GridSize: " << gridSize << endl << "Unroll: " << unroll << endl;
    cout << "Setup done!" << endl;
    //endregion

    //region Generation
    cout << "Fractal generation in process..." << endl;
    multibrot_kernel<<<gridSize, blockSize>>>(unroll,
                                              imageDevice,
                                              width, height, ratio,
                                              exponent, iterations, R, eps,
                                              borderR, borderG, borderB, borderThickness,
                                              normOrbitSkip, normLightIntensity, normLightAngle, normLightHeight,
                                              bgR, bgG, bgB,
                                              kR, kG, kB, kD,
                                              internalBorderR, internalBorderG, internalBorderB,
                                              internalCoreR, internalCoreG, internalCoreB, internalK,
                                              stripeDensity, stripeLightIntensity,
                                              zoom, posX, posY);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(imageHost, imageDevice, 4 * size * sizeof(unsigned char), hipMemcpyDeviceToHost));
    cout << "Generation done!" << endl;

    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, multibrot_kernel, blockSize, 0);
    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);
    double occupancy = (maxActiveBlocks * blockSize / props.warpSize) /
                       (double) (props.maxThreadsPerMultiProcessor / props.warpSize);
    cout << std::setprecision(4) << "Theoretical occupancy: " << occupancy << "%" << endl;
    //endregion

    for (int i = 0; i < size; i++) {
        rgb[i * 3 + 2] = imageHost[i * 4];
        rgb[i * 3 + 1] = imageHost[i * 4 + 1];
        rgb[i * 3] = imageHost[i * 4 + 2];
    }

    //region Cleanup
    free(imageHost);
    hipFree(imageDevice);
    hipDeviceReset();
    //endregion
    hipProfilerStop();
}
